#include "hip/hip_runtime.h"
#include "mandelbrot_config.h"
#include <hip/hip_runtime.h>
#include <>
#include <assert.h>
#include <SFML/Graphics.hpp>

#if 0

__global__ void mandelbrot_kernel(sf::Uint8* pixels)
{
    int screen_x = blockIdx.x * blockDim.x + threadIdx.x;
    int screen_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (screen_x >= WINDOW_WIDTH || screen_y >= WINDOW_HEIGHT)
        return;

    float pixel_step_x = 2.0f / WINDOW_WIDTH;
    float pixel_step_y = 2.0f / WINDOW_HEIGHT;

    float current_x_coord = -1.0f + screen_x * pixel_step_x;
    float current_y_coord = -1.0f + screen_y * pixel_step_y;

    int iterations = 0;
    float z_x  = 0.0f,
          z_y  = 0.0f,
          z_x2 = 0.0f,
          z_y2 = 0.0f;

    while (z_x2 + z_y2 < MAX_RADIUS_2 && iterations < MAX_ITERATION_DEPTH) {
        z_y = 2 * z_x * z_y + current_y_coord;
        z_x = z_x2 - z_y2 + current_x_coord;
        z_x2 = z_x * z_x;
        z_y2 = z_y * z_y;
        iterations++;
    }

    // ... Записываем цвета (пропущено для краткости)
}

void mandelbrot_cuda(sf::Uint8* pixels, float magnifier, float shiftX)
{
    shiftX -= 0.5f;
    magnifier -= 0.3f;

    dim3 blockSize(16, 16);
    dim3 gridSize((WINDOW_WIDTH + blockSize.x - 1) / blockSize.x, (WINDOW_HEIGHT + blockSize.y - 1) / blockSize.y);
    
    mandelbrot_kernel<<<gridSize, blockSize>>>(pixels, magnifier, shiftX);
}

int main(int argc, char* argv[])
{
    if (strcmp(argv[1], "cuda") != 0)
        assert(0);

    int nTests = 0;
    sscanf(argv[2], "%d", &nTests);

    sf::Uint8* pixels = nullptr; 
    hipMalloc(&pixels, WINDOW_WIDTH * WINDOW_HEIGHT * 4 * sizeof(sf::Uint8));

    for (int i = 0; i < nTests; i++)
        mandelbrot_cuda(pixels, 1.0f, 0.0f);
}

#endif

__global__ void mandelbrot_kernel(sf::Uint8* pixels, float magnifier, float shiftX)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= WINDOW_WIDTH || y >= WINDOW_HEIGHT)
        return;
    
    const float MAX_RADIUS_2 = MAX_RADIUS * MAX_RADIUS;
    const float ASPECT_RATIO = (float)(WINDOW_WIDTH) / WINDOW_HEIGHT;
    const float inv_magnifier = 1.0f / magnifier;
    const float colorScale = 255.0f / MAX_ITERATION_DEPTH;

    float pixel_step_x = ASPECT_RATIO * inv_magnifier * (2.0f / WINDOW_WIDTH);
    float pixel_step_y = inv_magnifier * (2.0f / WINDOW_HEIGHT);

    float current_x_coord = shiftX - ASPECT_RATIO * inv_magnifier + x * pixel_step_x;
    float current_y_coord = -1.0f * inv_magnifier + y * pixel_step_y;

    int iterations = 0;
    float z_x = 0.0f, z_y = 0.0f, z_x2 = 0.0f, z_y2 = 0.0f;

    while (z_x2 + z_y2 < MAX_RADIUS_2 && iterations < MAX_ITERATION_DEPTH) {
        z_y = 2 * z_x * z_y + current_y_coord;
        z_x = z_x2 - z_y2 + current_x_coord;
        z_x2 = z_x * z_x;
        z_y2 = z_y * z_y;
        iterations++;
    }

    sf::Uint8 r = 0, g = 0, b = 0;
    if (iterations < MAX_ITERATION_DEPTH) {
        float iterNormalized = iterations * colorScale;
        r = (sf::Uint8)(iterNormalized / 2);
        g = (sf::Uint8)(iterNormalized * 2 + 2);
        b = (sf::Uint8)(iterNormalized * 2 + 5);
    }

    int pixelIndex = (y * WINDOW_WIDTH + x) * 4;
    pixels[pixelIndex + 0] = r;
    pixels[pixelIndex + 1] = g;
    pixels[pixelIndex + 2] = b;
    pixels[pixelIndex + 3] = 255;
}

void mandelbrot_cuda(sf::Uint8* pixels, float magnifier, float shiftX) {
    shiftX -= 0.5f;
    magnifier -= 0.3f;
    
    sf::Uint8* d_pixels;
    size_t size = WINDOW_WIDTH * WINDOW_HEIGHT * 4 * sizeof(sf::Uint8);
    hipMalloc(&d_pixels, size);
    
    dim3 blockSize(16, 16);
    dim3 gridSize((WINDOW_WIDTH + blockSize.x - 1) / blockSize.x, (WINDOW_HEIGHT + blockSize.y - 1) / blockSize.y);
    
    mandelbrot_kernel<<<gridSize, blockSize>>>(d_pixels, magnifier, shiftX);
    hipMemcpy(pixels, d_pixels, size, hipMemcpyDeviceToHost);
    hipFree(d_pixels);
}

void mandelbrot_cuda_no_cpy(sf::Uint8* pixels, float magnifier, float shiftX) {
    shiftX -= 0.5f;
    magnifier -= 0.3f;

    dim3 blockSize(16, 16);
    dim3 gridSize((WINDOW_WIDTH  + blockSize.x - 1) / blockSize.x,
                  (WINDOW_HEIGHT + blockSize.y - 1) / blockSize.y);
    
    mandelbrot_kernel<<<gridSize, blockSize>>>(pixels, magnifier, shiftX);
}

#if 0

int main(int argc, char* argv[])
{
    if (strcmp(argv[1], "cuda") != 0)
        assert(0);

    int nTests = 0;
    sscanf(argv[2], "%d", &nTests);

    sf::Uint8* pixels = nullptr; 
    hipMalloc(&pixels, WINDOW_WIDTH * WINDOW_HEIGHT * 4 * sizeof(sf::Uint8));

    for (int i = 0; i < nTests; i++)
        mandelbrot_cuda(pixels, 1.0f, 0.0f);
}

#endif
