#include "hip/hip_runtime.h"
#include "mandelbrot_config.h"
#include <hip/hip_runtime.h>
#include <>
#include <assert.h>
#include <SFML/Graphics.hpp>

__global__ void mandelbrot_kernel(sf::Uint8* pixels, float magnifier, float shiftX)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= WINDOW_WIDTH || y >= WINDOW_HEIGHT)
        return;
    
    const float inv_magnifier = 1.0f / magnifier;
    const float COLOR_SCALE = 255.0f / MAX_ITERATION_DEPTH;

    float pixel_step_x = ASPECT_RATIO * inv_magnifier * (2.0f / WINDOW_WIDTH);
    float pixel_step_y = inv_magnifier * (2.0f / WINDOW_HEIGHT);

    float current_x_coord = shiftX - ASPECT_RATIO * inv_magnifier + x * pixel_step_x;
    float current_y_coord = -1.0f * inv_magnifier + y * pixel_step_y;

    int iterations = 0;
    float z_x = 0.0f, z_y = 0.0f, z_x2 = 0.0f, z_y2 = 0.0f;

    while (z_x2 + z_y2 < MAX_RADIUS_2 && iterations < MAX_ITERATION_DEPTH) {
        z_y = 2 * z_x * z_y + current_y_coord;
        z_x = z_x2 - z_y2 + current_x_coord;
        z_x2 = z_x * z_x;
        z_y2 = z_y * z_y;
        iterations++;
    }

    sf::Uint8 r = 0, g = 0, b = 0;
    if (iterations < MAX_ITERATION_DEPTH) {
        float iterNormalized = iterations * COLOR_SCALE;
        r = (sf::Uint8)(iterNormalized / 2);
        g = (sf::Uint8)(iterNormalized * 2 + 2);
        b = (sf::Uint8)(iterNormalized * 2 + 5);
    }

    int pixelIndex = (y * WINDOW_WIDTH + x) * 4;
    pixels[pixelIndex + 0] = r;
    pixels[pixelIndex + 1] = g;
    pixels[pixelIndex + 2] = b;
    pixels[pixelIndex + 3] = 255;
}

void mandelbrot_cuda(sf::Uint8* pixels, float magnifier, float shiftX) {
    shiftX    += SHIFT_X_OFFSET;
    magnifier += MAGNIFIER_OFFSET;
    
    sf::Uint8* d_pixels;
    size_t size = WINDOW_WIDTH * WINDOW_HEIGHT * 4 * sizeof(sf::Uint8);
    hipMalloc(&d_pixels, size);
    
    dim3 blockSize(16, 16);
    dim3 gridSize((WINDOW_WIDTH + blockSize.x - 1) / blockSize.x, (WINDOW_HEIGHT + blockSize.y - 1) / blockSize.y);
    
    mandelbrot_kernel<<<gridSize, blockSize>>>(d_pixels, magnifier, shiftX);
    hipMemcpy(pixels, d_pixels, size, hipMemcpyDeviceToHost);
    hipFree(d_pixels);
}

void mandelbrot_cuda_no_cpy(sf::Uint8* pixels, float magnifier, float shiftX) {
    shiftX -= 0.5f;
    magnifier -= 0.3f;

    dim3 blockSize(16, 16);
    dim3 gridSize((WINDOW_WIDTH  + blockSize.x - 1) / blockSize.x,
                  (WINDOW_HEIGHT + blockSize.y - 1) / blockSize.y);
    
    mandelbrot_kernel<<<gridSize, blockSize>>>(pixels, magnifier, shiftX);
}
